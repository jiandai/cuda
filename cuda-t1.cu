// ver 20170218 by jian: follow https://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/ except on unix direclty
// local nvcc: /.../.../cuda/7.5.18/x86_64-linux-2.6-rhel6/bin/nvcc
// example1.cpp : Defines the entry point for the console application.
//
 
//#include "stdafx.h" // for VC++ only
 
#include <stdio.h>
#include <hip/hip_runtime.h>
 
// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}
 
// main routine that executes on the host
int main(void)
{
  float *a_h, *a_d;  // Pointer to host & device arrays
  const int N = 10;  // Number of elements in arrays
  size_t size = N * sizeof(float);
  a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) a_h[i] = (float)i;
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  // Do calculation on device:
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  square_array <<< n_blocks, block_size >>> (a_d, N);
  // Retrieve result from device and store it in host array
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  // Print results
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  free(a_h); hipFree(a_d);
}