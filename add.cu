// ver 20170219 by jian
// ref: http://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
//cudaMalloc(), cudaFree(), cudaMemcpy()
//malloc(), free(), memcpy()
// concept of block, and thread

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int *a, int *b, int *c, int n) {
	//c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
	//c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
	int index=threadIdx.x+blockIdx.x*blockDim.x;
	if (index<n) c[index] = a[index] + b[index];
}

#include <cstdlib>

void random_ints(int* a, int m){
   for (int i = 0; i < m; ++i) a[i] = rand();
}

//# define N (2048*2048)
# define N (32*32)
# define THREADS_PER_BLOCK 16

int main (void) {
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	int size=N*sizeof(int);
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	//add<<<N,1>>>(d_a,d_b,d_c);
	//add<<<1,N>>>(d_a,d_b,d_c);
	//add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c,N);
	add<<<(N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c,N);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	for (int i=0;i<N;i++) {
		printf("%d: %d + %d = %d\n",i,a[i],b[i],c[i]);
	}
	free(a);free(b);free(c);
	hipFree(d_a);hipFree(d_b);hipFree(d_c);
	return 0;
}
